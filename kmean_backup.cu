#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>

#ifndef FLT_MAX
#define FLT_MAX 3.40282347e+38
#endif

__device__ inline float euclid_dist_2(int tid, int numObjects, int numAttributes, const float* __restrict__ attributes,
    int clusterId, const float* __restrict__ clusters){
    float ans=0.0;
    for(int i = 0; i < numAttributes; i++){
        float diff = attributes[tid + i*numObjects] - clusters[i + clusterId*numAttributes];
        ans += diff*diff;
    }

    return ans;
}


__device__ inline int find_nearest_point(int tid, \
    int numObjects,
    int     numAttributes,
    const float* __restrict__ attributess,
    const float* __restrict__ centers,         
    int     ncenters)
{
    int index = 0;
    float min_dist = FLT_MAX;

    for (int i = 0; i < ncenters; i++) {
        float dist;
        dist = euclid_dist_2(tid, numObjects, numAttributes, attributess, i, centers);  
        if (dist < min_dist) {
            min_dist = dist;
            index    = i;
        }
    }

    return(index);
}

__global__ void findNewClusterIndex(int numObjects, int numAttributes, const float* __restrict__ attributes, \
    int numClusters, const float* __restrict__ clusters, int* __restrict__ membership, \
    int* __restrict__ new_centers_len, float* __restrict__ new_centers, float* __restrict__ delta){
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if(tid < numObjects){
                /* find the index of nestest cluster centers */
        int index = find_nearest_point(tid,  numObjects, numAttributes, attributes, clusters, numClusters);
                /* if membership changes, increase delta by 1 */
        if (membership[tid] != index) atomicAdd(delta, 1.0f);

                /* assign the membership to object i */
        membership[tid] = index;

                /* update new cluster centers : sum of objects located within */
        atomicAdd(new_centers_len+index, 1);
    }
}

__global__ void updateNewCluster(int numObjects, int numAttributes, const float* __restrict__ attributes, \
    int numClusters, const float* __restrict__ clusters, int* __restrict__ membership, \
    int* __restrict__ new_centers_len, float* __restrict__ new_centers, float* __restrict__ delta){

    extern __shared__ float s[]; //__shared__

    int tid = blockDim.x*blockIdx.x + threadIdx.x;

    for(int i = threadIdx.x; i < numClusters*numAttributes; i += blockDim.x){
        s[i] = 0.0f;
    }

    __syncthreads();

    if(tid < numObjects){
        int index = membership[tid];
        for(int j = 0; j < numAttributes; j++){ 
            atomicAdd(s+index*numAttributes+j, attributes[tid + numObjects*j]);
        }
    }

    __syncthreads();

    for(int i = 0; i < numClusters; i++){
        for(int j = threadIdx.x; j < numAttributes; j += blockDim.x){ 
            atomicAdd(new_centers+i*numAttributes+j, s[j + numAttributes*i]);
        }
    }
}

__global__ void updateCenter(int numClusters, int numAttributes, float* __restrict__ clusters,\
    int* __restrict__ new_centers_len, float* __restrict__ new_centers){
        /* replace old cluster centers with new_centers */
    for(int i = blockIdx.x; i < numClusters; i += gridDim.x) {
        for(int j = threadIdx.x; j < numAttributes; j += blockDim.x) {
            if (new_centers_len[i] > 0)
                clusters[i*numAttributes + j] = new_centers[i*numAttributes + j] / new_centers_len[i];
                        new_centers[i*numAttributes + j] = 0.0;   /* set back to 0 */
        }
                new_centers_len[i] = 0;   /* set back to 0 */
    }
}

/*----< kmeans_clustering() >---------------------------------------------*/
void kmeans_clustering(int     numObjects,
 int     numAttributes,
                                                 float *attributes,    /* in: [numObjects][numAttributes] */
 int    *membership,
 int     numClusters,
 float*  clusters,
 float   threshold){

        int     *d_new_centers_len; /* [numClusters]: no. of points in each cluster */
    hipMalloc((void**)&d_new_centers_len, numClusters*sizeof(int));

    float  *d_new_centers;     /* [numClusters][numAttributes] */
    hipMalloc((void**)&d_new_centers, numClusters*numAttributes*sizeof(int));

    float    *d_delta;
    hipMalloc((void**)&d_delta, sizeof(float));

    float *d_attributes;
    hipMalloc((void**)&d_attributes, numObjects*numAttributes*sizeof(float));
    hipMemcpy(d_attributes, attributes, numObjects*numAttributes*sizeof(float), hipMemcpyDefault);

    int *d_membership;
    hipMalloc((void**)&d_membership, numObjects*sizeof(int));
    hipMemcpy(d_membership, membership, numObjects*sizeof(int), hipMemcpyDefault);

    float *d_clusters;
    hipMalloc((void**)&d_clusters, numClusters*numAttributes*sizeof(float));
    hipMemcpy(d_clusters, clusters, numClusters*numAttributes*sizeof(float), hipMemcpyDefault);

    float delta = 0.0;

    do {
        hipMemset(d_new_centers_len, 0, numClusters*sizeof(int));
        hipMemset(d_new_centers, 0, numClusters*numAttributes*sizeof(int));
        hipMemset(d_delta, 0, sizeof(int));

        int blockSize = 256;
        int gridSize = (numObjects+blockSize-1)/blockSize;
        findNewClusterIndex<<<gridSize, blockSize>>>(numObjects, numAttributes, d_attributes, numClusters, d_clusters, d_membership, d_new_centers_len, d_new_centers, d_delta);

        updateNewCluster<<<gridSize, blockSize, numClusters*numAttributes*sizeof(float)>>>(numObjects, numAttributes, d_attributes, numClusters, d_clusters, d_membership, d_new_centers_len, d_new_centers, d_delta);

        updateCenter<<<numClusters, blockSize>>>(numClusters, numAttributes, d_clusters, d_new_centers_len, d_new_centers);

        //delta /= numObjects;
        hipMemcpy(&delta, d_delta, sizeof(float), hipMemcpyDefault);

        printf("%.3f %.3f\n", delta, threshold);
    } while (delta > threshold);

    hipMemcpy(clusters, d_clusters, numClusters*numAttributes*sizeof(float), hipMemcpyDefault);
    hipMemcpy(membership,d_membership,numObjects*sizeof(int),hipMemcpyDefault);

    hipFree(d_new_centers_len);
    hipFree(d_new_centers);
    hipFree(d_delta);
    hipFree(d_attributes);
    hipFree(d_membership);
    hipFree(d_clusters);
}

int main(int argc, char **argv) {
    FILE * fin=fopen("oData.txt","r");
    FILE * fout=fopen("output.txt","w");
    const int maxd[] = {188,158,171,158,165,168,188,166,222,183,187,163,162,170,188,197,220,180,187,176,171,162,177,211,187,159,187,165,165,162,161,164,188,170,176,165,172,189,180,173,222,169,187,163,169,168,180,185,220,173,187,167,172,163,177,193,187,161,187,171,174,183,166,167,188,159,187,172,166,171,172,173,222,169,187,171,167,165,177,193,220,169,187,169,171,168,177,188,184,160,170,164,180,179,179,176,188,165,180,176,166,162,166,164,222,184,180,172,165,163,176,204,220,191,180,163,170,187,184,197,187,155,164,165,165,165,177,168};
    int nums = 4000000,dim = 128,k = 96;
    //sscanf(argv[1],"%d",&k);
    printf("k=%d\n",k);
    int *membership,*countM;
    float *data,*cluster,tmp;
    srand((unsigned)time(NULL));
    membership = (int *)calloc(nums,sizeof(int));
    data = (float *)calloc(nums*dim,sizeof(float));
    cluster = (float *)calloc(k*dim,sizeof(float));
    countM = (int *)calloc(k,sizeof(int));
    for (int i=0; i<nums; i++)
        for (int j=0; j<dim; j++) {
            fscanf(fin,"%f",&tmp);
            data[i+j*nums] = tmp;
        }
	//for (int i=0; i<nums*dim; i++) printf("%f ",data[i]);

        for (int i=0; i<nums; i++) {
            membership[i] = 0;
        }

        for (int i=0; i<k; i++) 
            for (int j=0; j<dim; j++) {
            //cluster[i*dim+j] = rand()%maxd[j];
                if (rand()%2 == 0) {
                    cluster[i*dim+j] = 0;
                }
                else {
                    cluster[i*dim+j] = maxd[j];
                }
            }

            kmeans_clustering(nums,dim,data,membership,k,cluster,1000);

            for (int i=0; i<k*dim; i++) {
                if  (i%dim == 0) fprintf(fout,"\n\n");
                fprintf(fout,"%f ",cluster[i]);
            }
            fprintf(fout,"\n\n-----------------------------------------\n\n");
            memset(countM,k*sizeof(int),0);
            for (int i=0; i<nums; i++) {
                fprintf(fout,"%d ",membership[i]);
                countM[membership[i]]++;
            }
            fprintf(fout,"\n\n-----------------------------------------\n\n");

            for (int i=0; i<k; i++) {
                fprintf(fout,"%d ",countM[i]);
            }
            fprintf(fout,"\n\n-----------------------------------------\n\n");

            free(countM);
            free(membership);
            free(data);
            free(cluster);
            fclose(fin);
            fclose(fout);
            return 0;
        }
