#include "hip/hip_runtime.h"
/*
    Latest verison of kmean with dynamic group number constrain.
*/

#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>

#ifndef FLT_MAX
#define FLT_MAX 3.40282347e+38
#endif

__device__ inline float euclid_dist_2(int tid, int numObjects, int numAttributes, const float* __restrict__ attributes,
    int clusterId, const float* __restrict__ clusters){
    float ans=0.0;
    for(int i = 0; i < numAttributes; i++){
        float diff = attributes[tid + i*numObjects] - clusters[i + clusterId*numAttributes];
        ans += diff*diff;
    }

    return ans;
}


__device__ inline int find_nearest_point(int tid, \
    int numObjects,
    int     numAttributes,
    const float* __restrict__ attributess,
    const float* __restrict__ centers,         
    int     ncenters,
    int* __restrict__ flag)
{
    int index = 0;
    float min_dist = FLT_MAX;

    for (int i = 0; i < ncenters; i++)
        if (flag[i] == 0) {
            float dist;
            dist = euclid_dist_2(tid, numObjects, numAttributes, attributess, i, centers);  
            if (dist < min_dist) {
                min_dist = dist;
                index    = i;
            }
        }

        return(index);
    }

    __global__ void findNewClusterIndex(int numObjects, int numAttributes, const float* __restrict__ attributes, \
        int numClusters, const float* __restrict__ clusters, int* __restrict__ membership, \
        int* __restrict__ new_centers_len, float* __restrict__ new_centers, float* __restrict__ delta,\
        int* __restrict__ flag) {
        int tid = blockIdx.x*blockDim.x + threadIdx.x;

        if(tid < numObjects){
                /* find the index of nestest cluster centers */
            int index = find_nearest_point(tid,  numObjects, numAttributes, attributes, clusters, numClusters, flag);
                /* if membership changes, increase delta by 1 */
            if (membership[tid] != index) atomicAdd(delta, 1.0f);

                /* assign the membership to object i */
            membership[tid] = index;

                /* update new cluster centers : sum of objects located within */
            atomicAdd(new_centers_len+index, 1);
        }
    }

    __global__ void updateNewCluster(int numObjects, int numAttributes, const float* __restrict__ attributes, \
        int numClusters, const float* __restrict__ clusters, int* __restrict__ membership, \
        int* __restrict__ new_centers_len, float* __restrict__ new_centers, float* __restrict__ delta){

        int tid = blockDim.x*blockIdx.x + threadIdx.x;

        if(tid < numObjects){
            int index = membership[tid];
            for(int j = 0; j < numAttributes; j++){ 
                atomicAdd(new_centers+index*numAttributes+j, attributes[tid + numObjects*j]);
            }
        }

        __syncthreads();

    }

    __global__ void updateCenter(int threshold, int numClusters, int numAttributes, float* __restrict__ clusters,\
        int* __restrict__ new_centers_len, float* __restrict__ new_centers,int* __restrict__ flag){
        /* replace old cluster centers with new_centers */
        for(int i = blockIdx.x; i < numClusters; i += gridDim.x) {
            for(int j = threadIdx.x; j < numAttributes; j += blockDim.x) {
                if (new_centers_len[i] > 0)
                    clusters[i*numAttributes + j] = new_centers[i*numAttributes + j] / new_centers_len[i];
        new_centers[i*numAttributes + j] = 0.0;   /* set back to 0 */
            }
    // new update
            if (new_centers_len[i] < threshold) flag[i] = 1;
    new_centers_len[i] = 0;   /* set back to 0 */
        }
        __syncthreads();
    }

/*----< kmeans_clustering() >---------------------------------------------*/
    void kmeans_clustering(int     numObjects,
       int     numAttributes,
                                                 float *attributes,    /* in: [numObjects][numAttributes] */
       int    *membership,
       int     numClusters,
       float*  clusters,
       float   threshold){

int     *d_new_centers_len; /* [numClusters]: no. of points in each cluster */
        hipMalloc((void**)&d_new_centers_len, numClusters*sizeof(int));

float  *d_new_centers;     /* [numClusters][numAttributes] */
        hipMalloc((void**)&d_new_centers, numClusters*numAttributes*sizeof(int));

        float    *d_delta;
        hipMalloc((void**)&d_delta, sizeof(float));

        float *d_attributes;
        hipMalloc((void**)&d_attributes, numObjects*numAttributes*sizeof(float));
        hipMemcpy(d_attributes, attributes, numObjects*numAttributes*sizeof(float), hipMemcpyDefault);

        int *d_membership;
        hipMalloc((void**)&d_membership, numObjects*sizeof(int));
        hipMemcpy(d_membership, membership, numObjects*sizeof(int), hipMemcpyDefault);

        float *d_clusters;
        hipMalloc((void**)&d_clusters, numClusters*numAttributes*sizeof(float));
        hipMemcpy(d_clusters, clusters, numClusters*numAttributes*sizeof(float), hipMemcpyDefault);

        float delta = 0.0;

        int *flag;
        hipMalloc((void**)&flag, numClusters*sizeof(int));
        hipMemset(flag, 0, numClusters*sizeof(int));

        int *s;
        s = (int *)calloc(numClusters,sizeof(int));

        int turns = 0,changeHold = 1;

        do {
            hipMemset(d_new_centers_len, 0, numClusters*sizeof(int));
            hipMemset(d_new_centers, 0, numClusters*numAttributes*sizeof(int));
            hipMemset(d_delta, 0, sizeof(int));


            int blockSize = 256;
            int gridSize = (numObjects+blockSize-1)/blockSize;
            findNewClusterIndex<<<gridSize, blockSize>>>(numObjects, numAttributes, d_attributes, numClusters, d_clusters, d_membership, d_new_centers_len, d_new_centers, d_delta,flag);

    // updateNewCluster<<<gridSize, blockSize, numClusters*numAttributes*sizeof(float)>>>(numObjects, numAttributes, d_attributes, numClusters, d_clusters, d_membership, d_new_centers_len, d_new_centers, d_delta);
            updateNewCluster<<<gridSize, blockSize>>>(numObjects, numAttributes, d_attributes, numClusters, d_clusters, d_membership, d_new_centers_len, d_new_centers, d_delta);

            updateCenter<<<numClusters, blockSize>>>(changeHold, numClusters, numAttributes, d_clusters, d_new_centers_len, d_new_centers, flag);

    //delta /= numObjects;
            hipMemcpy(&delta, d_delta, sizeof(float), hipMemcpyDefault);

            turns++;
            if (turns < 30) {
             if (turns%10 == 0) changeHold += 10;
         } else {
          if (turns%30 == 0 && changeHold < 50) changeHold += 30;
      }
      printf("%d %d %.3f %.3f\n",turns, changeHold, delta, threshold);

    // hipMemcpy(s, flag, numClusters*sizeof(int), hipMemcpyDefault);
    // for (int i=0; i<numClusters; i++) {
    //     printf("%d ",s[i]);
    // }
    // printf("\n\n");

  } while (delta > threshold);

  hipMemcpy(clusters, d_clusters, numClusters*numAttributes*sizeof(float), hipMemcpyDefault);
  hipMemcpy(membership,d_membership,numObjects*sizeof(int),hipMemcpyDefault);

  hipFree(d_new_centers_len);
  hipFree(d_new_centers);
  hipFree(d_delta);
  hipFree(d_attributes);
  hipFree(d_membership);
  hipFree(d_clusters);
  hipFree(flag);
  free(s);
}

int main(int argc, char **argv) {
    FILE * fin=fopen("oData.txt","r");
    FILE * outC=fopen("outCenter.txt","w");
    FILE * fout=fopen("output.txt","w");
    int nums = 4000000,dim = 128,k = 96;
    int thold = 500;
    sscanf(argv[1],"%d",&k);
    sscanf(argv[2],"%d",&nums);
    printf("k=%d\n",k);
    printf("n=%d\n",nums);
    int *membership,*countM;
    float *data,*cluster,tmp;
    srand((unsigned)time(NULL));
    membership = (int *)calloc(nums,sizeof(int));
    data = (float *)calloc(nums*dim,sizeof(float));
    cluster = (float *)calloc(k*dim,sizeof(float));
    countM = (int *)calloc(k,sizeof(int));
    for (int i=0; i<nums; i++)
        for (int j=0; j<dim; j++) {
            fscanf(fin,"%f",&tmp);
            data[i+j*nums] = tmp;
        }
	//for (int i=0; i<nums*dim; i++) printf("%f ",data[i]);

        for (int i=0; i<nums; i++) {
            membership[i] = 0;
        }
        int target;
        for (int i=0; i<k; i++) 
            for (int j=0; j<dim; j++) {
            //cluster[i*dim+j] = rand()%maxd[j];
               target = rand()%nums;
               cluster[i*dim+j]=data[target+j*nums];
           }

           kmeans_clustering(nums,dim,data,membership,k,cluster,thold);

           for (int i=0; i<k*dim; i++) {
             if  (i%dim == 0 && i>0) fprintf(outC,"\n");
             fprintf(outC,"%f ",cluster[i]);
         }
         memset(countM,k*sizeof(int),0);
         for (int i=0; i<nums; i++) {
            fprintf(fout,"%d ",membership[i]);
            countM[membership[i]]++;
        }
        fprintf(fout,"\n\n-----------------------------------------\n\n");

        for (int i=0; i<k; i++) {
            fprintf(fout,"%d ",countM[i]);
        }

        free(countM);
        free(membership);
        free(data);
        free(cluster);
        fclose(fin);
        fclose(fout);
        fclose(outC);
        return 0;
    }

